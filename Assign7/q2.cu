
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include<sys/time.h>

#define NUM 10000000

#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                             printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                             exit(-1);\
                                    }\
                             }while(0);
#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))

struct num_array{
                    int Xn;
};



__device__ void myXOR(struct num_array *a, int x,int y)
{
    int ans = x^y;
    a->Xn = ans;
    return;
}
__global__ void calculate(char *mem, char *result, int num)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i >= num)
		return;
	
	struct num_array *b = (struct num_array *)(result);
	struct num_array *a = (struct num_array *)(mem + (i*sizeof(int)));
	if(b->Xn==(int)INFINITY)
	{
		b->Xn=a->Xn;
	}
	
	else
	{
		int x = b->Xn;	
		int y= a->Xn;
		myXOR(a,x,y);
	}
}

int main(int argc, char **argv)
{
	struct timeval start, end, t_start, t_end;
	int i;
	struct num_array *pa;
	char *ptr;
	char *ans_ptr;
	char *sptr;
	char *gpu_mem;
	char *gpu_ans;	   
	unsigned long num = NUM;   /*Default value of num from MACRO*/
	int blocks;

	if(argc == 2)
	{
		num = atoi(argv[1]);   /*Update after checking*/
		if(num <= 0)
			num = NUM;
	}

	/* Allocate host (CPU) memory and initialize*/

	ptr = (char *)malloc(num * sizeof(int));
	ans_ptr=(char *)malloc(sizeof(int));
	sptr = ptr; 
	for(i=0; i<num; ++i)
	{
		pa = (struct num_array *) sptr;
		pa->Xn = random();
		sptr += sizeof(int);
	}
    
	pa=(struct num_array *) ans_ptr;
	pa->Xn=(int)INFINITY;
    
	gettimeofday(&t_start, NULL);
    
	/* Allocate GPU memory and copy from CPU --> GPU*/

	hipMalloc(&gpu_mem, num*sizeof(int));
	CUDA_ERROR_EXIT("hipMalloc");

	hipMalloc(&gpu_ans,sizeof(int));
	CUDA_ERROR_EXIT("hipMalloc");

	hipMemcpy(gpu_mem, ptr, num*sizeof(int) , hipMemcpyHostToDevice);
	CUDA_ERROR_EXIT("hipMemcpy");

	hipMemcpy(gpu_ans, ans_ptr,sizeof(int) , hipMemcpyHostToDevice);
	CUDA_ERROR_EXIT("hipMemcpy");
    
	gettimeofday(&start, NULL);
    
	blocks = num /1024;
    
	if(num % 1024)
		++blocks;

	calculate<<<blocks, 1024>>>(gpu_mem,gpu_ans,num);
	CUDA_ERROR_EXIT("kernel invocation");
	gettimeofday(&end, NULL);
    
	/* Copy back result*/

	
	hipMemcpy(ans_ptr, gpu_ans,sizeof(int) , hipMemcpyDeviceToHost);
	CUDA_ERROR_EXIT("memcpy");

	gettimeofday(&t_end, NULL);
    
	printf("Total time = %ld microsecs Processsing =%ld microsecs\n", TDIFF(t_start, t_end), TDIFF(start, end));
	hipFree(gpu_mem);
	hipFree(gpu_ans);
   
	/*Print result*/ 
	pa=(struct num_array *) ans_ptr;
	printf("Result=%d\n",pa->Xn);

    
	free(ptr);
	free(ans_ptr);
}
